#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <chrono>
#include <hip/hip_fp16.h>
#include <hipblas.h>

#define N 16384  // Matrix size

__global__ void helloFromGPU() {
    printf("Hello from GPU!\n");
}

__global__ void matrixMultiply(float *A, float *B, float *C) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        float sum = 0;
        for (int i = 0; i < N; i++) {
            sum += A[row * N + i]*B[i * N + col];
        }
        C[row * N + col] = sum;
    }
}

int main() {
    float *d_A, *d_B, *d_C;  // Device copies of A, B, C
    int size = N * N * sizeof(float);

    // Cublas
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Allocate memory
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // Initialize host copies of A, B
    float *A = (float *)malloc(size);
    float *B = (float *)malloc(size);
    float *C = (float *)malloc(size);

    for (int i = 0; i < 1; i++) {
        for (int i = 0; i < N*N; i++) {
            A[i] = static_cast<float>(rand()) / RAND_MAX;
            B[i] = static_cast<float>(rand()) / RAND_MAX;
        }

        auto start = std::chrono::high_resolution_clock::now();
        // Copy inputs to device
        hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
        hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
        // cudaDeviceSynchronize();
        auto stop = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
        printf("To cuda: %ld micros\n", duration.count());


        // Enable TF32 tensor core math mode
        hipblasSetMathMode(handle, HIPBLAS_TF32_TENSOR_OP_MATH);
    
        float alpha = 1.0f;
        float beta = 0.0f;

        start = std::chrono::high_resolution_clock::now();
        // Launch kernel on the GPU
        //cublasSgemm(handle, CUBLAS_OP_N, CUBLAS_OP_N, N, N, N, &alpha, d_A, N, d_B, N, &beta, d_C, N);
        hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_A, HIP_R_32F, N, d_B, HIP_R_32F, N, &beta, d_C, HIP_R_32F, N, HIP_R_32F, HIPBLAS_GEMM_DEFAULT);
        hipDeviceSynchronize();
        stop = std::chrono::high_resolution_clock::now();
        duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
        printf("Multiply cublas: %ld micros\n", duration.count());
        

        start = std::chrono::high_resolution_clock::now();
        // Launch kernel on the GPU
        dim3 threadsPerBlock(32, 32, 1);
        dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y);
        matrixMultiply<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C);
        hipDeviceSynchronize();
        stop = std::chrono::high_resolution_clock::now();
        duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
        printf("Multiply mine: %ld micros\n", duration.count());

        hipError_t error = hipGetLastError();
        if (error != hipSuccess) {
            std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
        } else {
            std::cout << "Kernel launch successful!" << std::endl;
        }

        start = std::chrono::high_resolution_clock::now();
        // Copy result back to host
        hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        stop = std::chrono::high_resolution_clock::now();
        duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
        printf("To cpu: %ld micros\n", duration.count());
    }

    // //Print result
    // for (int i = 0; i < N * N; i++) {
    //     printf("%f   ", C[i]);
    //     if ((i + 1) % N == 0) printf("\n");
    // }

    // Cleanup
    free(A); free(B); free(C);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    hipblasDestroy(handle);

    return 0;
}

